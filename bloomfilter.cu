#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define get_random rand()

#define M 100000000
#define max_str_length 8

using namespace std;

// Header code starts here

__device__ int strlen(char *str){
    int c;
    for (c = 0; str[c] != '\0'; c++);
    return c;
}

__device__ void string_to_bytes(char* str, int *bytes){
    int n = strlen(str);
    unsigned char byte;
    int i, j, c = 0;

    for (i = 0; i < n ; i++){
        for (j = 7; j >= 0; j--) {
            byte = (str[i] >> j) & 1;
            bytes[c++] = byte;
        }
    }
}



__device__ int fnv1s(char* str){

    int FNVPRIME = 0x01000193;
    int FNVINIT = 0x811c9dc5;

    int bytes[100];
    string_to_bytes(str, bytes);
    int hash = FNVINIT;
    for (int i = 0; i < strlen(str)*8; i++){
        hash *= FNVPRIME;
        hash ^= bytes[i];
    }

    return abs(hash);
}

__device__ int hashmix(char* str, int a, int b) {
    int bytes[100];
    string_to_bytes(str, bytes);
    int c = bytes[0];

    for (int i = 1; i < strlen(str)*8; i++){
        a -= (b + c);  a ^= (c >> 13);
        b -= (c + a);  b ^= (a << 8);
        c -= (a + b);  c ^= (b >> 13);
        a -= (b + c);  a ^= (c >> 12);
        b -= (c + a);  b ^= (a << 16);
        c -= (a + b);  c ^= (b >> 5);
        a -= (b + c);  a ^= (c >> 3);
        b -= (c + a);  b ^= (a << 10);
        c -= (a + b);  c ^= (b >> 15);
        c ^= bytes[i];
    }

    return abs(c);
}

__device__ int murmur (char* key, int seed)
{
    int m = 0x5bd1e995;

    int len = strlen(key);
    int h = seed ^ len;

    char * data = key;

    switch(len)
    {
        case 4: h ^= data[3] << 24;
        case 3: h ^= data[2] << 16;
        case 2: h ^= data[1] << 8;
        case 1: h ^= data[0];
            h *= m;
    };

    h ^= h >> 13;
    h *= m;
    h ^= h >> 15;

    return h;
}


__device__ unsigned long
djb2(char *str)
{
    unsigned long hash = 5381;
    int c;

    while (c = *str++)
        hash = ((hash << 5) + hash) + c; /* hash * 33 + c */

    return hash;
}


// Header code ends here

void print_bit_array(char *bitarray)
{
    for (int i=0; i<M; i++)
        if(bitarray[i] == 1)
            printf("%d ", i);
    printf("\n");
}

__global__ void insert_parallel(char *, char *, int, int, int);

__global__ void lookup_parallel(char *, char *, int, int, int, char *);

int main(int argc, char *argv[])
{

    FILE *fp_insert, *fp_lookup;
    int num_inserts, num_lookups, i;

    if (argc != 3)
    {
        printf("Usage: ./bloomfilter <inserts_filename> <lookups_filename>\n");
        exit(0);
    }

    char *insert_filename = argv[1];
    char *lookup_filename = argv[2];

    fp_insert = fopen(insert_filename, "r");
    fp_lookup = fopen(lookup_filename, "r");

    if (fp_insert == NULL)
    {
        printf("Not a valid insert file\n");
        exit(0);
    }

    if (fp_lookup == NULL)
    {
        printf("Not a valid lookup file\n");
        exit(0);
    }

    fscanf(fp_insert, "%d", &num_inserts);

    char *inserts = (char *) malloc(num_inserts*max_str_length*sizeof(char));
//    char inserts[num_inserts][max_str_length];


    for (i=0; i<num_inserts; i++)
    {
        fscanf(fp_insert, "%s", &inserts[i*max_str_length]);
    }



    fclose(fp_insert);

    fscanf(fp_lookup, "%d", &num_lookups);
//    char lookups[num_lookups][max_str_length];
    char *lookups = (char *) malloc(num_lookups*max_str_length*sizeof(char));

    for (i=0; i<num_lookups; i++)
    {
//        printf("%d ", i);
        fscanf(fp_lookup, "%s", &lookups[i*max_str_length]);
    }



//    for (int i = 0; i < 1000; i++)
//        printf("%s ", lookups[i*max_str_length]);

    fclose(fp_lookup);

    srand(42);
    int seed1 = get_random;
    int seed2 = get_random;

    char *c_inserts = NULL;
    if (hipMalloc((void**)&c_inserts, num_inserts*max_str_length*sizeof(char)) != hipSuccess ) {
        printf("Error while allocating memory for insert array");
        exit(1);
    }

    char *c_bits = NULL;
    if (hipMalloc((void**)&c_bits, M*sizeof(char)) != hipSuccess ) {
        printf("Error while allocating memory for bit array");
        exit(1);
    }

    hipMemset(c_bits, 0, M*sizeof(char));

    char *c_maybe = NULL;
    if (hipMalloc((void**)&c_maybe, num_lookups*sizeof(char)) != hipSuccess ) {
        printf("Error while allocating memory for maybe array");
        exit(1);
    }

    char *c_lookups = NULL;
    if (hipMalloc((void**)&c_lookups, num_lookups*max_str_length*sizeof(char)) != hipSuccess ) {
        printf("Error while allocating memory for insert array");
        exit(1);
    }

    if (hipMemcpy(c_inserts, inserts, num_inserts*max_str_length*sizeof(char), hipMemcpyHostToDevice) != hipSuccess) {
        printf("Error while copying to device from Host for insert array");
        exit(1);
    }

    if (hipMemcpy(c_lookups, lookups, num_lookups*max_str_length*sizeof(char), hipMemcpyHostToDevice) != hipSuccess) {
        printf("Error while copying to device from Host for lookup array");
        exit(1);
    }

    int no_of_block = (int)(num_inserts/64) + 1;
    insert_parallel<<<no_of_block, 64>>>(c_inserts, c_bits, num_inserts, seed1, seed2);
    hipDeviceSynchronize();


    int no_of_block_lookups = (int)(num_lookups/64) + 1;
    lookup_parallel<<<no_of_block_lookups, 64>>>(c_lookups, c_bits, num_lookups, seed1, seed2, c_maybe);
    hipDeviceSynchronize();

    char *maybe = (char *) malloc(num_lookups*sizeof(char));

    hipMemcpy(maybe, c_maybe, num_lookups*sizeof(char), hipMemcpyDeviceToHost);

    int c1 = 0, c2 = 0;
    for (int i = 0; i < num_lookups; i++){
        if (maybe[i]) c1++;
        else c2++;
    }

    cout << c1 << " " << c2<<endl;

    return 0;
}

__global__ void insert_parallel(char *inserts, char *bits, int size, int seed1, int seed2){
    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size){
        int p = hashmix(&inserts[i*max_str_length], seed1, seed2) % M;
        int w = djb2(&inserts[i*max_str_length]) % M;
//        int w = murmur(&inserts[i*max_str_length], seed1) % M;
        int r = fnv1s(&inserts[i*max_str_length]) % M;
        bits[p] = 1;
        bits[w] = 1;
        bits[r] = 1;
//        printf("%d - %c, ", w, bits[w]+48);
    }
}

__global__ void lookup_parallel(char *inserts, char *bits, int size, int seed1, int seed2, char *maybe){
    unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size){

        int p = hashmix(&inserts[i*max_str_length], seed1, seed2) % M;
        int w = djb2(&inserts[i*max_str_length]) % M;
//        int w = murmur(&inserts[i*max_str_length], seed1) % M;
        int r = fnv1s(&inserts[i*max_str_length]) % M;
        if (bits[p] == 1 && bits[w] == 1 && bits[r] == 1) maybe[i] = 1;
        else maybe[i] = 0;
//            if (bits[w] == 1) maybe[i] = 1;
//            else maybe[i] = 0;
    }
}